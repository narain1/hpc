#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

__global__ void dropout(float *input, float *output, int size, float dropout_rate, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float random = hiprand_uniform(&state);
        output[idx] = (random > dropout_rate) ? input[idx] : 0;
    }
}

int main () {
    int size = 1024;
    float *input, *output;
    float dropout_rate = 0.1;
    unsigned long long seed = 1234;
    hipMallocManaged(&input, size * sizeof(float));
    hipMallocManaged(&output, size * sizeof(float));
    for (int i = 0; i < size; i++) {
        input[i] = rand() / (float)RAND_MAX;
    }
    dropout<<<(size + 255) / 256, 256>>>(input, output, size, dropout_rate, seed);
    hipDeviceSynchronize();
    for (int i = 0; i < 25; i++) {
        printf("%f\n", output[i]);
    }
    hipFree(input);
    hipFree(output);
    return 0;
}