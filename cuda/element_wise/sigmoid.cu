#include <hip/hip_runtime.h>

#include <float.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <stdlib.h>

#define WARPSIZE 32

__global__
void sigmoid_f32_kernel(float *a, float *b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)  {
        b[idx] = 1.0f / (1.0f + expf(-a[idx]));
    }
}

__global__
void sigmoid_f16_kernel(half *a, half *b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)  {
        b[idx] = 1.0f / (1.0f + expf(-__half2float(a[idx])));
    }
}

void rand_init(float *a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    int n = 16000;
    float *a = (float *)malloc(n * sizeof(float));
    float *out = (float *)malloc(n * sizeof(float));
    rand_init(a, n);
    float *d_a, *d_out;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    sigmoid_f32_kernel<<<(n + WARPSIZE - 1) / WARPSIZE, WARPSIZE>>>(d_a, d_out, n);
    hipMemcpy(a, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 5; i++) {
        printf("%f ", a[i]);
    }

    printf("\n");
    
    half *a_h = (half *)malloc(n * sizeof(half));
    half *out_h = (half *)malloc(n * sizeof(half));
    for (int i = 0; i < 5; i++) {
        a_h[i] = __float2half(a[i]);
    }
    half *d_a_h, *d_out_h;
    hipMalloc(&d_a_h, n * sizeof(half));
    hipMalloc(&d_out_h, n * sizeof(half));
    hipMemcpy(d_a_h, a_h, n * sizeof(half), hipMemcpyHostToDevice);
    sigmoid_f16_kernel<<<(n + WARPSIZE - 1) / WARPSIZE, WARPSIZE>>>(d_a_h, d_out_h, n);
    hipMemcpy(out_h, d_out_h, n * sizeof(half), hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++) {
        printf("%f ", __half2float(out_h[i]));
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_a_h);
    hipFree(d_out_h);
    free(a);
    free(out);
    free(a_h);
    free(out_h);
    return 0;

}