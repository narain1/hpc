#include <hip/hip_runtime.h>


__global__
void convolution_1d(float *n, float *m, float *r, int n_size, int m_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float p_value = 0.0f;
    if (i < n_size) {
        r[i] = 0;
        for (int j = 0; j < m_size; j++) {
            if (i - j >= 0) {
                p_value += n[i - j] * m[j];
            }
        }
        r[i] = p_value;
    }
}

void convolution(float *n, float *m, float *r, int n_size, int m_size) {
    float *d_n, *d_m, *d_r;
    hipMalloc(&d_n, n_size * sizeof(float));
    hipMalloc(&d_m, m_size * sizeof(float));
    hipMalloc(&d_r, n_size * sizeof(float));

    hipMemcpy(d_n, n, n_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, m, m_size * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (n_size + block_size - 1) / block_size;

    convolution_1d<<<num_blocks, block_size>>>(d_n, d_m, d_r, n_size, m_size);

    hipMemcpy(r, d_r, n_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_n);
    hipFree(d_m);
    hipFree(d_r);
}

// cudaMemcpyToSymbol