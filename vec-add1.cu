#include <hip/hip_runtime.h>
#include <iostream>


__global__ void vector_add(float *a, float *b, float *c, int n, int m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=idx; i < n * m; i += stride) {
        int row = i / m;
        c[i] = a[i] + b[row]; // global memory access
    }
}

int main() {
    int n = 1024;
    int m = 512;

    float *a = new float[n * m];
    float *b = new float[n];
    float *c = new float[n * m];

    for (int i=0; i<n*m; i++) {
        a[i] = static_cast<float>(i);
    }
    for (int i=0; i<n; i++) {
        b[i] = static_cast<float>(i);
    } 

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n*m * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n*m*sizeof(float));

    hipMemcpy(d_a, a, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int num_blocks = (n * m + blockSize - 1) / blockSize;

    vector_add<<<num_blocks, blockSize>>>(d_a, d_b, d_c, n, m);
    hipMemcpy(c, d_c, n * m * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] a;
    delete[] b;
    delete[] c;

}